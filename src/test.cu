

#include <hip/hip_runtime.h>
#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//


//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
__global__
void add(int *a, int *b, int N) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {

    int N = 1000;
    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //

    int * ha = (int *)malloc(sizeof(int)*1000);
    int * hb = (int *)malloc(sizeof(int)*1000);

    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
	hb[i] = -1;
    }

    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db, N);

    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("[%d]", hb[i]);
	if((i+1)%40 == 0) printf("\n");
    }

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}
